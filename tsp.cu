#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <limits.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

#define THREADS 512
#define MAXCITIES 1296

extern "C" int tsp(int, int,int, int, int, float *, float *);

__global__ void TspKernel(int kCities, int kSamples, float *kPosx, float *kPosy, int *dlength)
{
  __shared__ int local_length;
  register int iter, i, j, len, from, to;
  register float dx, dy;
  register unsigned short tmp;
  unsigned short tour[MAXCITIES+1];
  hiprandState rndstate;
  iter = threadIdx.x + blockIdx.x * blockDim.x;
  tour[kCities] = 0;
  local_length = INT_MAX;
 
  if(iter==0)
  {
     *dlength = INT_MAX;
  }
  __syncthreads();

/* iterate number of sample times */
  if (iter < kSamples) {
  
/* generate a random tour */
    hiprand_init(iter, 0, 0, &rndstate);
    for (i = 1; i < kCities; i++) tour[i] = i;
    for (i = 1; i < kCities; i++) {
      j = hiprand(&rndstate) % (kCities - 1) + 1;
      tmp = tour[i];
      tour[i] = tour[j];
      tour[j] = tmp;
    }

 /* compute tour length */
    len = 0;
    from = 0;
    for (i = 1; i <= kCities; i++) {
      to = tour[i];
      dx = kPosx[to] - kPosx[from];
      dy = kPosy[to] - kPosy[from];
      len += (int)(sqrtf(dx * dx + dy * dy) + 0.5f);
      from = to;
    }

 /* check if new shortest tour */
     atomicMin(&local_length, len);
  }
  __syncthreads();

  if (threadIdx.x == 0) {
    atomicMin(dlength, local_length);
 }
}

static int read_input(char *filename, float *posx, float *posy)
{
  register int cnt;
  int i1, cities;
  float i2, i3;
  register FILE *f;

 /* open input text file */
  f = fopen(filename, "r+t");
  if (f == NULL) {fprintf(stderr, "could not open file %s\n", filename); exit(-1);}

 /* read the number of cities from first line */
  cities = -1;
  fscanf(f, "%d\n", &cities);
  if ((cities < 1) || (cities >= MAXCITIES)) {fprintf(stderr, "cities out of range\n"); exit(-1);}

  /* read in the cities' coordinates */
  cnt = 0;
  while (fscanf(f, "%d %f %f\n", &i1, &i2, &i3)) {
            posx[cnt] = i2;
    posy[cnt] = i3;
    cnt++;
    if (cnt > cities) {fprintf(stderr, "input too long\n"); exit(-1);}
    if (cnt != i1) {fprintf(stderr, "input line mismatch\n"); exit(-1);}
  }
  if (cnt != cities) {fprintf(stderr, "wrong number of cities read\n"); exit(-1);}

  /* return the number of cities */
  fclose(f);
  return cities;
}

int main(int argc, char *argv[])
{
  register int blocks, samples, c_samples, o_samples, cities;
  float posx[MAXCITIES], posy[MAXCITIES], *dposx, *dposy;
  struct timeval start, end;
  int *dlength, length, o_length, final_length, thread_count;

  printf("TSP v1.0(CUDA)\n");

  /* check command line */
  if (argc != 4) {fprintf(stderr, "usage: %s input_file_name number_of_samples\n", argv[0]); exit(-1);}
  cities = read_input(argv[1], posx, posy);
  samples = atoi(argv[2]);
  if (samples < 1) {fprintf(stderr, "number of samples must be at least 1\n"); exit(-1);}
  printf("%d cities and %d samples (%s)\n", cities, samples, argv[1]);
  o_length = INT_MAX;
  
  thread_count = strtol(argv[3],NULL,10);
  c_samples = (int)ceil(samples/2);
  o_samples = (int)floor(samples/2);
  blocks = (c_samples + THREADS - 1) / THREADS;

  if (hipSuccess != hipMalloc((void **)&dlength, sizeof(int))) fprintf(stderr, "could not allocate array\n");
  if (hipSuccess != hipMalloc((void **)&dposx, (cities*sizeof(float)))) fprintf(stderr, "could not allocate array\n");
  if (hipSuccess != hipMalloc((void **)&dposy, (cities*sizeof(float)))) fprintf(stderr, "could not allocate array\n");

  /* start time */
  gettimeofday(&start, NULL);

  if (hipSuccess != hipMemcpy(dposx, posx, (cities*sizeof(float)), hipMemcpyHostToDevice)) fprintf(stderr, "copying of posx to device failed\n");
  if (hipSuccess != hipMemcpy(dposy, posy, (cities*sizeof(float)), hipMemcpyHostToDevice)) fprintf(stderr, "copying of posy to device failed\n");

  TspKernel<<<blocks, THREADS>>>(cities, c_samples, dposx, dposy, dlength);

  o_length = tsp(thread_count, samples, o_samples, cities, o_length, posx, posy);

 if (hipSuccess != hipMemcpy(&length, dlength, sizeof(1), hipMemcpyDeviceToHost)) fprintf(stderr, "copying of dlength from device failed\n");

 /* end time */
  gettimeofday(&end, NULL);
  printf("runtime: %.4lf s\n", end.tv_sec + end.tv_usec / 1000000.0 - start.tv_sec - start.tv_usec / 1000000.0);

 /* output result */
 if(length < o_length) final_length = length;
 else final_length = o_length;
  
  printf("length of shortest found tour: %d\n\n", final_length);

 /* freeing memory */
  hipFree(dlength);
  hipFree(dposx);
  hipFree(dposy);
  return 0;
}


